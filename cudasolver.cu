#include "hip/hip_runtime.h"
/*
 * Copyright 2018 Azlehria
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be included
 * in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#if defined __INTELLISENSE__
 /* reduce vstudio warnings (__byteperm, blockIdx...) */
#  define __CUDA_ARCH__ 1
#  include <>
#  undef __CUDA_ARCH__
#  define __HIPCC__ 1
#  include <device_atomic_functions.hpp>
#  undef __HIPCC__
#  include <cstring>
#endif //__INTELLISENSE__

using uint64_t = unsigned long long;

struct __align__(8) uchar8
{
  unsigned char s0, s1, s2, s3, s4, s5, s6, s7;
};

__constant__ uint64_t mid[25]{ 0 };
__constant__ uint64_t target{ 0 };
__constant__ uint64_t threads{ 0 };

__device__ uint64_t solution_count{ 0 };
__device__ uint64_t solutions[256]{ 0 };

__device__
auto ROTL64( uint64_t& output, uint64_t const x, uint64_t const y ) -> void
{
  output = (x << y) ^ (x >> (64u - y));
}

//__device__
//auto bswap_32( uint32_t& input ) -> void
//{
//  uchar4 temp{ reinterpret_cast<uchar4&>(input) };
//  input = temp.x << 24 | temp.y << 16 | temp.z << 8 | temp.w;
//}

__device__
auto bswap_64( uint64_t input ) -> uint64_t
{
  //uint2& t_ref{ reinterpret_cast<uint2&>(input) };
  //uchar4 &tempx{ reinterpret_cast<uchar4&>(t_ref.x) },
  //       tempy{ reinterpret_cast<uchar4&>(t_ref.y) };
  //t_ref.x = tempy.x << 24 | tempy.y << 16 | tempy.z << 8 | tempy.w;
  //t_ref.y = tempx.x << 24 | tempx.y << 16 | tempx.z << 8 | tempx.w;
  //return input;
  uchar8& t_ref{ reinterpret_cast<uchar8&>(input) };
  return uint64_t( t_ref.s0 << 24 | t_ref.s1 << 16 | t_ref.s2 << 8 | t_ref.s3 ) << 32 | t_ref.s4 << 24 | t_ref.s5 << 16 | t_ref.s6 << 8 | t_ref.s7;
}

__device__
auto xor5( uint64_t& output, uint64_t (&s)[25], uint64_t offset ) -> void
{
#if __CUDA_ARCH__
  asm( "{"
       "  xor.b64 %0, %1, %2;"
       "  xor.b64 %0, %0, %3;"
       "  xor.b64 %0, %0, %4;"
       "  xor.b64 %0, %0, %5;"
       "}"
       : "=l"(output)
       : "l"(s[0u + offset]), "l"(s[5u + offset]), "l"(s[10u + offset]), "l"(s[15u + offset]), "l"(s[20u + offset]) );
#else
  output = s[0u + offset] ^ s[5u + offset] ^ s[10u + offset] ^ s[15u + offset] ^ s[20u + offset];
#endif
}

__device__
auto xor3( uint64_t& a, uint64_t const b, uint64_t const c ) -> void
{
  a = a ^ b ^ c;
}

__device__
auto theta_parity( uint64_t (&t)[5], uint64_t (&s)[25] ) -> void
{
  xor5( t[1], s, 0u );
  xor5( t[2], s, 1u );
  xor5( t[3], s, 2u );
  xor5( t[4], s, 3u );
  xor5( t[0], s, 4u );
}

__device__
auto theta_xor( uint64_t (&s)[25], uint64_t offset, uint64_t const t, uint64_t const v ) -> void
{
  uint64_t u;
  ROTL64( u, v, 1u );
  xor3( s[ 0u + offset], u, t );
  xor3( s[ 5u + offset], u, t );
  xor3( s[10u + offset], u, t );
  xor3( s[15u + offset], u, t );
  xor3( s[20u + offset], u, t );
}

__device__
auto theta( uint64_t (&s)[25], uint64_t (&t)[5] ) -> void
{
  theta_parity( t, s );

  theta_xor( s, 0u, t[0], t[2] );
  theta_xor( s, 1u, t[1], t[3] );
  theta_xor( s, 2u, t[2], t[4] );
  theta_xor( s, 3u, t[3], t[0] );
  theta_xor( s, 4u, t[4], t[1] );
}

__device__
auto chi_group( uint64_t (&s)[25], uint64_t offset, uint64_t (&t)[5] ) -> void
{
  s[0u + offset] = t[0] ^ ~t[1] & t[2];
  s[1u + offset] = t[1] ^ ~t[2] & t[3];
  s[2u + offset] = t[2] ^ ~t[3] & t[4];
  s[3u + offset] = t[3] ^ ~t[4] & t[0];
  s[4u + offset] = t[4] ^ ~t[0] & t[1];
}

__device__
auto chi( uint64_t (&s)[25], uint64_t offset, uint64_t (&t)[5] ) -> void
{
  t[0] = s[0u + offset];
  t[1] = s[1u + offset];
  t[2] = s[2u + offset];
  t[3] = s[3u + offset];
  t[4] = s[4u + offset];
  chi_group( s, offset, t );
}

__device__
auto keccak_round( uint64_t (&s)[25], uint64_t (&t)[5], uint64_t const rc ) -> void
{
  theta( s, t );

  t[0] = s[1];
  ROTL64( s[ 1], s[ 6], 44u );
  ROTL64( s[ 6], s[ 9], 20u );
  ROTL64( s[ 9], s[22], 61u );
  ROTL64( s[22], s[14], 39u );
  ROTL64( s[14], s[20], 18u );
  ROTL64( s[20], s[ 2], 62u );
  ROTL64( s[ 2], s[12], 43u );
  ROTL64( s[12], s[13], 25u );
  ROTL64( s[13], s[19],  8u );
  ROTL64( s[19], s[23], 56u );
  ROTL64( s[23], s[15], 41u );
  ROTL64( s[15], s[ 4], 27u );
  ROTL64( s[ 4], s[24], 14u );
  ROTL64( s[24], s[21],  2u );
  ROTL64( s[21], s[ 8], 55u );
  ROTL64( s[ 8], s[16], 45u );
  ROTL64( s[16], s[ 5], 36u );
  ROTL64( s[ 5], s[ 3], 28u );
  ROTL64( s[ 3], s[18], 21u );
  ROTL64( s[18], s[17], 15u );
  ROTL64( s[17], s[11], 10u );
  ROTL64( s[11], s[ 7],  6u );
  ROTL64( s[ 7], s[10],  3u );
  ROTL64( s[10], t[ 0],  1u );

  chi( s,  0u, t );
  chi( s,  5u, t );
  chi( s, 10u, t );
  chi( s, 15u, t );
  chi( s, 20u, t );

  s[0] ^= rc;
}

__device__
auto keccak_first( uint64_t (&s)[25], uint64_t (&t)[5] ) -> void
{
  memcpy( s, mid, sizeof( s ) );
  uint64_t D{ 0u };
  uint64_t const nounce{ threads + (blockDim.x * blockIdx.x + threadIdx.x) };
  ROTL64( D, nounce, 44u ); // 44
  s[2] ^= D;
  ROTL64( D, nounce, 14u ); // 14
  s[4] ^= D;
  ROTL64( D, nounce, 20u ); // 20
  s[6] ^= D;
  ROTL64( D, nounce, 62u ); // 62
  s[9] ^= D;
  ROTL64( D, nounce, 7u ); // 7
  s[11] ^= D;
  ROTL64( D, nounce, 8u ); // 8
  s[13] ^= D;
  ROTL64( D, nounce, 27u ); // 27
  s[15] ^= D;
  ROTL64( D, nounce, 16u ); // 16
  s[18] ^= D;
  ROTL64( D, nounce, 63u ); //63
  s[20] ^= D;
  ROTL64( D, nounce, 55u ); // 55
  s[21] ^= D;
  ROTL64( D, nounce, 39u ); // 39
  s[22] ^= D;

  chi( s,  0u, t );
  chi( s,  5u, t );
  chi( s, 10u, t );
  chi( s, 15u, t );
  chi( s, 20u, t );

  s[0] ^= 0x0000000000000001ull;
}

extern "C"
__global__
void cuda_mine()
{
  uint64_t state[25], C[5];

  keccak_first( state, C );

  keccak_round( state, C, 0x0000000000008082uLL );
  keccak_round( state, C, 0x800000000000808auLL );
  keccak_round( state, C, 0x8000000080008000uLL );
  keccak_round( state, C, 0x000000000000808buLL );
  keccak_round( state, C, 0x0000000080000001uLL );
  keccak_round( state, C, 0x8000000080008081uLL );
  keccak_round( state, C, 0x8000000000008009uLL );
  keccak_round( state, C, 0x000000000000008auLL );
  keccak_round( state, C, 0x0000000000000088uLL );
  keccak_round( state, C, 0x0000000080008009uLL );
  keccak_round( state, C, 0x000000008000000auLL );
  keccak_round( state, C, 0x000000008000808buLL );
  keccak_round( state, C, 0x800000000000008buLL );
  keccak_round( state, C, 0x8000000000008089uLL );
  keccak_round( state, C, 0x8000000000008003uLL );
  keccak_round( state, C, 0x8000000000008002uLL );
  keccak_round( state, C, 0x8000000000000080uLL );
  keccak_round( state, C, 0x000000000000800auLL );
  keccak_round( state, C, 0x800000008000000auLL );
  keccak_round( state, C, 0x8000000080008081uLL );
  keccak_round( state, C, 0x8000000000008080uLL );
  keccak_round( state, C, 0x0000000080000001uLL );

  theta_parity( C, state );

  ROTL64( state[1], C[2], 1u );
  xor3( state[ 0], state[1], C[0] );
  ROTL64( state[1], C[3], 1u );
  xor3( state[ 6], state[1], C[1] );
  ROTL64( state[1], C[4], 1u );
  xor3( state[12], state[1], C[2] );

  ROTL64( state[2], state[6], 44u );
  ROTL64( state[3], state[12], 43u );

  state[1] = state[0] ^ ~state[2] & state[3];
  state[1] ^= 0x8000000080008008uLL;

  //if( reinterpret_cast<uint2&>(state[0]).x ) return;
  //bswap_64( state[0] );
  if( bswap_64( state[1] ) > target ) return;

  uint64_t cIdx{ atomicAdd( &solution_count, 1u ) };
  if( cIdx >= 256u ) return;

  solutions[cIdx] = threads + (blockDim.x * blockIdx.x + threadIdx.x);
}

// --------------------------------------------------------------------

//using vec64 = union
//{
//private:
//  uint64_t raw_;
//public:
//  struct
//  { uint32_t x, y; };
//  operator uint64_t()
//  { return raw_; }
//  operator uint2()
//  { return { x, y }; }
//  uint64_t operator=( uint64_t rt )
//  { return raw_ = rt; }
//  uint2 operator=( uint2 rt )
//  { return { x = rt.x, y = rt.y }; }
//};
