#include "hip/hip_runtime.h"
/*
Author: Mikers
date march 4, 2018 for 0xbitcoin dev

based off of https://github.com/Dunhili/SHA3-gpu-brute-force-cracker/blob/master/sha3.cu

 * Author: Brian Bowden
 * Date: 5/12/14
 *
 * This is the parallel version of SHA-3.
 */

#include <iostream>
#include <cstring>
#include "cudasolver.h"

#ifdef __INTELLISENSE__
 /* reduce vstudio warnings (__byteperm, blockIdx...) */
#  include <hip/device_functions.h>
#  include <>
#  include <hip/hip_runtime.h>
#  include <hip/hip_runtime.h>
#endif //__INTELLISENSE__

#define cudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__, m_device)

__host__ inline
auto __cudaSafeCall( hipError_t err, char const* file, int32_t const line, int32_t device_id ) -> void
{
#ifndef CUDA_NDEBUG
  if (hipSuccess != err) {
    std::cerr << "CUDA device " << device_id
              << " encountered an asynchronous error in file '" << file
              << "' in line " << line
              << " : " << hipGetErrorString( err ) << ".\n";
    hipError_t syncErr = hipGetLastError();
    if( syncErr )
    {
      std::cerr << "Synchronous error " << syncErr << ":"
                << hipGetErrorString( syncErr ) << " was also encountered.\n";
    }
    exit( EXIT_FAILURE );
  }
#endif
}

#if __CUDA_ARCH__ < 350
__device__ __constant__ const uint64_t RC[24] = {
  0x0000000000008082, 0x800000000000808a, 0x8000000080008000,
  0x000000000000808b, 0x0000000080000001, 0x8000000080008081,
  0x8000000000008009, 0x000000000000008a, 0x0000000000000088,
  0x0000000080008009, 0x000000008000000a, 0x000000008000808b,
  0x800000000000008b, 0x8000000000008089, 0x8000000000008003,
  0x8000000000008002, 0x8000000000000080, 0x000000000000800a,
  0x800000008000000a, 0x8000000080008081, 0x8000000000008080,
  0x0000000080000001
};
#endif
__constant__ uint64_t d_mid[25];
__constant__ uint64_t d_target;

__device__ __forceinline__
auto ROTL64( uint64_t& output, uint64_t const x, uint32_t const y ) -> void
{
  output = (x << y) ^ (x >> (64 - y));
}

__device__ __forceinline__
auto bswap_64( uint64_t const input ) -> uint64_t const
{
  uint64_t output;
  asm( "{"
       "  prmt.b32 %0, %3, 0, 0x0123;"
       "  prmt.b32 %1, %2, 0, 0x0123;"
       "}" : "=r"(reinterpret_cast<uint2&>(output).x), "=r"(reinterpret_cast<uint2&>(output).y)
           : "r"(reinterpret_cast<uint2 const&>(input).x), "r"(reinterpret_cast<uint2 const&>(input).y) );
  return output;
}

__device__ __forceinline__
auto xor5( uint64_t& output, uint64_t const* const s ) -> void
{
  asm( "{"
       "  xor.b64 %0, %1, %2;"
       "  xor.b64 %0, %0, %3;"
       "  xor.b64 %0, %0, %4;"
       "  xor.b64 %0, %0, %5;"
       "}" : "=l"( output ) : "l"( s[0] ), "l"( s[5] ), "l"( s[10] ), "l"( s[15] ), "l"( s[20] ) );
}

__device__ __forceinline__
auto xor3( uint64_t& a, uint64_t const b, uint64_t const c ) -> void
{
#if __CUDA_ARCH__ >= 500
  asm( "{"
       "  lop3.b32 %0, %0, %2, %4, 0x96;"
       "  lop3.b32 %1, %1, %3, %5, 0x96;"
       "}" : "+r"(reinterpret_cast<uint2&>(a).x), "+r"(reinterpret_cast<uint2&>(a).y)
           : "r"(reinterpret_cast<uint2 const&>(b).x), "r"(reinterpret_cast<uint2 const&>(b).y),
             "r"(reinterpret_cast<uint2 const&>(c).x), "r"(reinterpret_cast<uint2 const&>(c).y) );
#else
  asm( "{"
       "  xor.b64 %0, %0, %1;"
       "  xor.b64 %0, %0, %2;"
       "}" : "+l"(a) : "l"(b), "l"(c) );
#endif
}

__device__ __forceinline__
auto theta_parity( uint64_t* __restrict__ t, uint64_t const* const __restrict__ s ) -> void
{
  xor5( t[1], &s[0] );
  xor5( t[2], &s[1] );
  xor5( t[3], &s[2] );
  xor5( t[4], &s[3] );
  xor5( t[0], &s[4] );
}

__device__ __forceinline__
auto theta_xor( uint64_t* __restrict__ s, uint64_t const t, uint64_t& u, uint64_t const v ) -> void
{
  ROTL64( u, v, 1 );
  xor3( s[ 0], u, t );
  xor3( s[ 5], u, t );
  xor3( s[10], u, t );
  xor3( s[15], u, t );
  xor3( s[20], u, t );
}

__device__ __forceinline__
auto theta( uint64_t* __restrict__ s, uint64_t* __restrict__ t, uint64_t* __restrict__ u ) -> void
{
  theta_parity( t, s );

  theta_xor( &s[0], t[0], u[0], t[2] );
  theta_xor( &s[1], t[1], u[1], t[3] );
  theta_xor( &s[2], t[2], u[2], t[4] );
  theta_xor( &s[3], t[3], u[3], t[0] );
  theta_xor( &s[4], t[4], u[4], t[1] );
}

__device__ __forceinline__
auto chi_single( uint64_t& output, uint64_t const a, uint64_t const b, uint64_t const c ) -> void
{
#if __CUDA_ARCH__ >= 500
  asm( "{"
       "  lop3.b32 %0, %2, %4, %6, 0xD2;"
       "  lop3.b32 %1, %3, %5, %7, 0xD2;"
       "}" : "+r"(reinterpret_cast<uint2&>(output).x), "+r"(reinterpret_cast<uint2&>(output).y)
           : "r"(reinterpret_cast<uint2 const&>(a).x), "r"(reinterpret_cast<uint2 const&>(a).y),
             "r"(reinterpret_cast<uint2 const&>(b).x), "r"(reinterpret_cast<uint2 const&>(b).y),
             "r"(reinterpret_cast<uint2 const&>(c).x), "r"(reinterpret_cast<uint2 const&>(c).y) );
#else
  output = a ^ ((~b) & c);
#endif
}

__device__ __forceinline__
auto chi_group( uint64_t* __restrict__ s, uint64_t const* const __restrict__ t ) -> void
{
  chi_single( s[0], t[0], t[1], t[2] );
  chi_single( s[1], t[1], t[2], t[3] );
  chi_single( s[2], t[2], t[3], t[4] );
  chi_single( s[3], t[3], t[4], t[0] );
  chi_single( s[4], t[4], t[0], t[1] );
}

__device__ __forceinline__
auto chi( uint64_t* __restrict__ s, uint64_t* __restrict__ t ) -> void
{
  t[0] = s[0];
  t[1] = s[1];
  t[2] = s[2];
  t[3] = s[3];
  t[4] = s[4];
  chi_group( s, t );
}

__device__ __forceinline__
auto keccak_round( uint64_t* __restrict__ s, uint64_t* __restrict__ t, uint64_t* __restrict__ u, uint64_t const rc ) -> void
{
  theta( s, t, u );
  
  t[0] = s[1];
  ROTL64( s[ 1], s[ 6], 44 );
  ROTL64( s[ 6], s[ 9], 20 );
  ROTL64( s[ 9], s[22], 61 );
  ROTL64( s[22], s[14], 39 );
  ROTL64( s[14], s[20], 18 );
  ROTL64( s[20], s[ 2], 62 );
  ROTL64( s[ 2], s[12], 43 );
  ROTL64( s[12], s[13], 25 );
  ROTL64( s[13], s[19],  8 );
  ROTL64( s[19], s[23], 56 );
  ROTL64( s[23], s[15], 41 );
  ROTL64( s[15], s[ 4], 27 );
  ROTL64( s[ 4], s[24], 14 );
  ROTL64( s[24], s[21],  2 );
  ROTL64( s[21], s[ 8], 55 );
  ROTL64( s[ 8], s[16], 45 );
  ROTL64( s[16], s[ 5], 36 );
  ROTL64( s[ 5], s[ 3], 28 );
  ROTL64( s[ 3], s[18], 21 );
  ROTL64( s[18], s[17], 15 );
  ROTL64( s[17], s[11], 10 );
  ROTL64( s[11], s[ 7],  6 );
  ROTL64( s[ 7], s[10],  3 );
  ROTL64( s[10], t[ 0],  1 );

  chi( &s[ 0], t );
  chi( &s[ 5], t );
  chi( &s[10], t );
  chi( &s[15], t );
  chi( &s[20], t );

  s[0] ^= rc;
}

__device__ __forceinline__
auto keccak_first( uint64_t* __restrict__ s, uint64_t* __restrict__ t, uint64_t const nounce ) -> void
{
  uint64_t n[11]{ 0 };
  ROTL64( n[ 0], nounce, 7 );
  ROTL64( n[ 1], n[0],  1 );
  ROTL64( n[ 2], n[1],  6 );
  ROTL64( n[ 3], n[2],  2 );
  ROTL64( n[ 4], n[3],  4 );
  ROTL64( n[ 5], n[4],  7 );
  ROTL64( n[ 6], n[5], 12 );
  ROTL64( n[ 7], n[6],  5 );
  ROTL64( n[ 8], n[7], 11 );
  ROTL64( n[ 9], n[8],  7 );
  ROTL64( n[10], n[9],  1 );

  t[0] = d_mid[0];
  t[1] = d_mid[1];
  t[2] = d_mid[2] ^ n[7];
  t[3] = d_mid[3];
  t[4] = d_mid[4] ^ n[2];
  chi_group( &s[0], t );
  s[0] ^= 0x0000000000000001ull;

  t[0] = d_mid[5];
  t[1] = d_mid[6] ^ n[4];
  t[2] = d_mid[7];
  t[3] = d_mid[8];
  t[4] = d_mid[9] ^ n[9];
  chi_group( &s[5], t );

  t[0] = d_mid[10];
  t[1] = d_mid[11] ^ n[0];
  t[2] = d_mid[12];
  t[3] = d_mid[13] ^ n[1];
  t[4] = d_mid[14];
  chi_group( &s[10], t );

  t[0] = d_mid[15] ^ n[5];
  t[1] = d_mid[16];
  t[2] = d_mid[17];
  t[3] = d_mid[18] ^ n[3];
  t[4] = d_mid[19];
  chi_group( &s[15], t );

  t[0] = d_mid[20] ^ n[10];
  t[1] = d_mid[21] ^ n[8];
  t[2] = d_mid[22] ^ n[6];
  t[3] = d_mid[23];
  t[4] = d_mid[24];
  chi_group( &s[20], t );
}
__global__
void cuda_mine( uint64_t* __restrict__ solution, uint32_t* __restrict__ solution_count, uint64_t const threads )
{
  uint64_t const nounce{ threads + (blockDim.x * blockIdx.x + threadIdx.x) };

  uint64_t state[25], C[5], D[5];

  keccak_first( state, C, nounce );

#if __CUDA_ARCH__ >= 350
  keccak_round( state, C, D, 0x0000000000008082ull );
  keccak_round( state, C, D, 0x800000000000808aull );
  keccak_round( state, C, D, 0x8000000080008000ull );
  keccak_round( state, C, D, 0x000000000000808bull );
  keccak_round( state, C, D, 0x0000000080000001ull );
  keccak_round( state, C, D, 0x8000000080008081ull );
  keccak_round( state, C, D, 0x8000000000008009ull );
  keccak_round( state, C, D, 0x000000000000008aull );
  keccak_round( state, C, D, 0x0000000000000088ull );
  keccak_round( state, C, D, 0x0000000080008009ull );
  keccak_round( state, C, D, 0x000000008000000aull );
  keccak_round( state, C, D, 0x000000008000808bull );
  keccak_round( state, C, D, 0x800000000000008bull );
  keccak_round( state, C, D, 0x8000000000008089ull );
  keccak_round( state, C, D, 0x8000000000008003ull );
  keccak_round( state, C, D, 0x8000000000008002ull );
  keccak_round( state, C, D, 0x8000000000000080ull );
  keccak_round( state, C, D, 0x000000000000800aull );
  keccak_round( state, C, D, 0x800000008000000aull );
  keccak_round( state, C, D, 0x8000000080008081ull );
  keccak_round( state, C, D, 0x8000000000008080ull );
  keccak_round( state, C, D, 0x0000000080000001ull );
#else
  for( int32_t i = 0; i < 22; ++i )
  {
    keccak_round( state, C, D, RC[i] );
  }
#endif // __CUDA_ARCH__ >= 350

  theta_parity( C, state );

  ROTL64( D[0], C[2], 1 );
  ROTL64( D[1], C[3], 1 );
  ROTL64( D[2], C[4], 1 );

  xor3( state[ 0], D[0], C[0] );
  xor3( state[ 6], D[1], C[1] );
  xor3( state[12], D[2], C[2] );
  ROTL64( state[ 6], state[ 6], 44 );
  ROTL64( state[12], state[12], 43 );

  chi_single( state[0], state[0], state[6], state[12] );
  state[0] ^= 0x8000000080008008ull;

  if( bswap_64( state[0] ) <= d_target )
  {
    uint64_t cIdx{ atomicAdd( solution_count, 1 ) };
    if( cIdx >= 256 ) return;

    solution[cIdx] = nounce;
  }
}

// --------------------------------------------------------------------

auto CUDASolver::cudaInit() -> void
{
  hipSetDevice( m_device );

  hipDeviceProp_t device_prop;
  cudaSafeCall( hipGetDeviceProperties( &device_prop, m_device ) );

  int32_t compute_version{ device_prop.major * 100 + device_prop.minor * 10 };

  if( compute_version <= 500 )
  {
    m_intensity = m_intensity <= 40.55 ? m_intensity : 40.55;
    m_threads = static_cast<uint64_t>( std::pow( 2, m_intensity <= 40.55 ? m_intensity : 40.55 ) );
  }

  m_block.x = compute_version > 500 ? TPB50 : TPB35;
  m_grid.x = uint32_t((m_threads + m_block.x - 1) / m_block.x);

  if( !m_gpu_initialized )
  {
    // CPU usage goes _insane_ without this.
    cudaSafeCall( hipDeviceReset() );
    // so we don't actually _use_ L1 or local memory . . .
    cudaSafeCall( hipSetDeviceFlags( hipDeviceScheduleBlockingSync ) );

    cudaSafeCall( hipMalloc( reinterpret_cast<void**>(&d_solution_count), 4 ) );
    cudaSafeCall( hipHostMalloc( reinterpret_cast<void**>(&h_solution_count), 4 ) );
    cudaSafeCall( hipMalloc( reinterpret_cast<void**>(&d_solutions), 256*8 ) );
    cudaSafeCall( hipHostMalloc( reinterpret_cast<void**>(&h_solutions), 256*8 ) );

    cudaResetSolution();

    m_gpu_initialized = true;
  }
}

auto CUDASolver::cudaCleanup() -> void
{
  cudaSafeCall( hipSetDevice( m_device ) );

  cudaSafeCall( hipDeviceSynchronize() );

  cudaSafeCall( hipFree( d_solution_count ) );
  cudaSafeCall( hipHostFree( h_solution_count ) );
  cudaSafeCall( hipFree( d_solutions ) );
  cudaSafeCall( hipHostFree( h_solutions ) );

  cudaSafeCall( hipDeviceReset() );

  m_gpu_initialized = false;
}

auto CUDASolver::cudaResetSolution() -> void
{
  hipSetDevice( m_device );

  std::memset( h_solution_count, 0u, 4 );
  cudaSafeCall( hipMemset( d_solution_count, 0u, 4 ) );
}

auto CUDASolver::pushTarget() -> void
{
  hipSetDevice( m_device );

  uint64_t target{ getTarget() };
  cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( d_target), &target, 8, 0, hipMemcpyHostToDevice) );

  m_new_target = false;
}

auto CUDASolver::pushMessage() -> void
{
  hipSetDevice( m_device );

  cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( d_mid), getMidstate().data(), 200, 0, hipMemcpyHostToDevice) );

  m_new_message = false;
}

auto CUDASolver::findSolution() -> void
{
  cudaInit();

  hipSetDevice( m_device );

  do
  {
    if( m_new_target ) { pushTarget(); }
    if( m_new_message ) { pushMessage(); }

    cuda_mine <<< m_grid, m_block >>> ( d_solutions, d_solution_count, getNextSearchSpace() );
    hipError_t syncErr = hipGetLastError();
    hipError_t asyncErr = hipDeviceSynchronize();
    if( syncErr | asyncErr != hipSuccess )
    {
      if( syncErr )
      {
        std::cerr << "Kernel launch encountered synchronous error "
                  << syncErr
                  << ": \x1b[38;5;196m"
                  << hipGetErrorString( syncErr )
                  << ".\x1b[0m\n"
                  << "Check your hardware configuration.\n";
      }
      if( asyncErr )
      {
        std::cerr << "Kernel launch encountered asynchronous error "
                  << asyncErr
                  << ": \x1b[38;5;196m"
                  << hipGetErrorString( asyncErr )
                  << ".\x1b[0m\n"
                  << "Check your hardware configuration.\n";
      }
      exit( EXIT_FAILURE );
    }

    cudaSafeCall( hipMemcpy( h_solution_count, d_solution_count, 4, hipMemcpyDeviceToHost ) );

    if( *h_solution_count )
    {
      cudaSafeCall( hipMemcpy( h_solutions, d_solutions, (*h_solution_count)*8, hipMemcpyDeviceToHost ) );
      pushSolutions();
      cudaResetSolution();
    }
  } while( !m_stop );

  m_stopped = true;
}
